#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <memory.h>

#define CHECK(call)													\
{																	\
const hipError_t error = call;										\
if(error != hipSuccess)											\
{																	\
	printf("Error: %s:%d, ", __FILE__, __LINE__);					\
	printf("CODE:%d, reason:%s\n", error, hipGetErrorString(error));\
	exit(1);														\
}																	\
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0e-8;
	bool match = 1;
	for(int i = 0; i < N; i++) {
		if (abs(hostRef[i] - gpuRef[i] > epsilon)) {
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
			break;
		}
	}
	if (match) printf("Arrays match, \n\n");
}

void initialData(float *ip, int size) {
	time_t t;
	srand((unsigned)time(&t));

	for (int i = 0; i < size; i++) {
		ip[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
	for (int idx = 0; idx < N; idx++)
	{
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C)
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

int main(int argc, char**argv)
{
	printf("%s Starting...\n", argv[0]);

	int dev = 0;
	hipSetDevice(dev);  // set up device;

	int nElem = 32;
	size_t nBytes = nElem * sizeof(float);

	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	// initialize data at host side
	initialData(h_A, nElem);
	initialData(h_B, nElem);

	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// malloc device global memory
	float *d_A, *d_B, *d_C;
	hipMalloc((float**)&d_A, nBytes);
	hipMalloc((float**)&d_B, nBytes);
	hipMalloc((float**)&d_C, nBytes);

	// transfer data from host to device
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

	// invoke kernel
	dim3 block(nElem);
	dim3 grid((nElem + block.x - 1) / nElem);

	sumArraysOnGPU <<<grid, block >>> (d_A, d_B, d_C);
	// copy kernel result back to host size
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	// add vector at host side
	sumArraysOnHost(h_A, h_B, hostRef, nElem);

	checkResult(hostRef, gpuRef, nElem);

	// free
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	return 0;
}